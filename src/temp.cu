#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

// This code assumes that your device support block size of 1024
#define MAX_RANGE 9999

#define funcCheck(stmt) do {                                                    \
        hipError_t err = stmt;                                               \
        if (err != hipSuccess) {                                             \
            printf( "Failed to run stmt %d ", __LINE__);                       \
            printf( "Got CUDA error ...  %s ", hipGetErrorString(err));    \
            return -1;                                                        \
        }                                                                     \
    } while(0)

// Compute C = A * B
__global__ void matrixMultiplyShared(float * A, float * B, float * C,
                                    int numARows, int numAColumns,
                                    int numBRows, int numBColumns,
                                    int numCRows, int numCColumns) 
{
    __shared__ float sA[32][32];   // Tile size of 32x32 
    __shared__ float sB[32][32];

    int Row = blockDim.y*blockIdx.y + threadIdx.y;
    int Col = blockDim.x*blockIdx.x + threadIdx.x;
    float Cvalue = 0.0;
    sA[threadIdx.y][threadIdx.x] = 0.0;
    sB[threadIdx.y][threadIdx.x] = 0.0;

    for (int k = 0; k < (((numAColumns - 1)/ 32) + 1); k++)
    {
        if ( (Row < numARows) && (threadIdx.x + (k*32)) < numAColumns)
        {
            sA[threadIdx.y][threadIdx.x] = A[(Row*numAColumns) + threadIdx.x + (k*32)];
        }
        else
        {
            sA[threadIdx.y][threadIdx.x] = 0.0;
        }            
        if ( Col < numBColumns && (threadIdx.y + k*32) < numBRows)
        {
            sB[threadIdx.y][threadIdx.x] = B[(threadIdx.y + k*32)*numBColumns + Col];
        }
        else
        {
            sB[threadIdx.y][threadIdx.x] = 0.0;
        }            
        __syncthreads();

        for (int j = 0; j < 32; ++j)
        {
            Cvalue += sA[threadIdx.y][j] * sB[j][threadIdx.x];
        }
    }
    if (Row < numCRows && Col < numCColumns)
    {
        C[Row*numCColumns + Col] = Cvalue;
    }
}

void matMultiplyOnHost(float * A, float * B, float * C, int numARows,
                        int numAColumns, int numBRows, int numBColumns,
                        int numCRows, int numCColumns)
{
    for (int i=0; i < numARows; i ++)
    {
        for (int j = 0; j < numAColumns; j++)
        {
            C[i*numCColumns + j ] = 0.0;
            for (int k = 0; k < numCColumns; k++)
            {
                C[i*numCColumns + j ] += A[i*numAColumns + k] * B [k*numBColumns + j];
            }
        }
    }
    return;
}

int main(int argc, char ** argv) {
    float * hostA; // The A matrix
    float * hostB; // The B matrix
    float * hostC; // The output C matrix
    float * hostComputedC;
    float * deviceA;
    float * deviceB;
    float * deviceC;

    // Please adjust rows and columns according to you need.
    int numARows = 512; // number of rows in the matrix A
    int numAColumns = 512; // number of columns in the matrix A
    int numBRows = 512; // number of rows in the matrix B
    int numBColumns = 512; // number of columns in the matrix B

    int numCRows; // number of rows in the matrix C (you have to set this)
    int numCColumns; // number of columns in the matrix C (you have to set this)

    hostA = (float *) malloc(sizeof(float)*numARows*numAColumns);
    hostB = (float *) malloc(sizeof(float)*numBRows*numBColumns);

    for (int i = 0; i < numARows*numAColumns; i++)
    {
        hostA[i] = (rand() % MAX_RANGE) / 2.0;
    }
    for (int i = 0; i < numBRows*numBColumns; i++)
    {
        hostB[i] = (rand() % MAX_RANGE) / 2.0;
    }

    // Setting numCRows and numCColumns
    numCRows = numARows;
    numCColumns = numBColumns;

    hostC = (float *) malloc(sizeof(float)*numCRows*numCColumns);    
    hostComputedC = (float *) malloc(sizeof(float)*numCRows*numCColumns);    

    // Allocating GPU memory
    funcCheck(hipMalloc((void **)&deviceA, sizeof(float)*numARows*numAColumns));
    funcCheck(hipMalloc((void **)&deviceB, sizeof(float)*numBRows*numBColumns));
    funcCheck(hipMalloc((void **)&deviceC, sizeof(float)*numCRows*numCColumns));

    // Copy memory to the GPU 
    funcCheck(hipMemcpy(deviceA, hostA, sizeof(float)*numARows*numAColumns, hipMemcpyHostToDevice));
    funcCheck(hipMemcpy(deviceB, hostB, sizeof(float)*numBRows*numBColumns, hipMemcpyHostToDevice));

    // Initialize the grid and block dimensions 
    dim3 dimBlock(32, 32, 1);    
    dim3 dimGrid((numCColumns/32) + 1, (numCRows/32) + 1, 1);

    //@@ Launch the GPU Kernel here
    matrixMultiplyShared<<<dimGrid, dimBlock>>>(deviceA, deviceB, deviceC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);    

    hipError_t err1 = hipPeekAtLastError();
    hipDeviceSynchronize();
    printf( "Got CUDA error ... %s \n", hipGetErrorString(err1));

    // Copy the results in GPU memory back to the CPU    
    funcCheck(hipMemcpy(hostC, deviceC, sizeof(float)*numCRows*numCColumns, hipMemcpyDeviceToHost));

    matMultiplyOnHost(hostA, hostB, hostComputedC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);

    for (int i=0; i < numCColumns*numCRows; i++)
    {
        if (hostComputedC[i]  != hostC[i] )
        {
            printf("Mismatch at Row = %d Col = %d hostComputed[] = %f --device[] %f\n", i / numCColumns, i % numCColumns, hostComputedC[i], hostC[i]);
            break;
        }
    }
    // Free the GPU memory
    funcCheck(hipFree(deviceA));
    funcCheck(hipFree(deviceB));        
    funcCheck(hipFree(deviceC));    

    free(hostA);
    free(hostB);
    free(hostC);
    free(hostComputedC);

    return 0;
}