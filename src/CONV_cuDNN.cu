#include "CONV_cuDNN.cuh"

using namespace std;

void convolution_cudnn(float* h_input, TensorDim inputDims, float* h_kernels, TensorDim KernelDims, float* h_output)
{ 
  GpuTimer timer;
  hipdnnHandle_t cudnn;
  hipdnnCreate(&cudnn);
 
  hipdnnTensorDescriptor_t input_descriptor;
  checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
  checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor, /*format=*/HIPDNN_TENSOR_NCHW, /*dataType=*/HIPDNN_DATA_FLOAT, /*batch_size=*/inputDims.n,
                                        /*channels=*/inputDims.c, /*image_height=*/inputDims.h, /*image_width=*/inputDims.w));

  hipdnnTensorDescriptor_t output_descriptor;
  checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
  checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor, /*format=*/HIPDNN_TENSOR_NCHW, /*dataType=*/HIPDNN_DATA_FLOAT, /*batch_size=*/inputDims.n,
                                        /*channels=*/KernelDims.n, /*image_height=*/inputDims.h, /*image_width=*/inputDims.w));

  hipdnnFilterDescriptor_t kernel_descriptor;
  checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
  checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor, /*dataType=*/HIPDNN_DATA_FLOAT, /*format=*/HIPDNN_TENSOR_NCHW, /*out_channels=*/KernelDims.n,
                                        /*in_channels=*/inputDims.c, /*kernel_height=*/KernelDims.h, /*kernel_width=*/KernelDims.w));


  hipdnnConvolutionDescriptor_t convolution_descriptor;
  checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
  checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor, /*pad_height=*/1, /*pad_width=*/1, /*vertical_stride=*/1, /*horizontal_stride=*/1,
                                           /*dilation_height=*/1, /*dilation_width=*/1, /*mode=*/HIPDNN_CROSS_CORRELATION, /*computeType=*/HIPDNN_DATA_FLOAT));


  hipdnnConvolutionFwdAlgo_t convolution_algorithm;
  checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(cudnn, input_descriptor, kernel_descriptor, convolution_descriptor, output_descriptor,
                                                 HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, /*memoryLimitInBytes=*/0, &convolution_algorithm));


  size_t workspace_bytes = 0;
  checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn, input_descriptor, kernel_descriptor, convolution_descriptor,
                                                   output_descriptor, convolution_algorithm, &workspace_bytes));
 
  void* d_workspace{nullptr};
  hipMalloc(&d_workspace, workspace_bytes);

  int in_tensor_bytes = inputDims.n * inputDims.h * inputDims.w * inputDims.c * sizeof(float);

  float* d_input{nullptr};
  hipMalloc(&d_input, in_tensor_bytes);

  int out_tensor_bytes = inputDims.n * inputDims.h * inputDims.w  * KernelDims.n * sizeof(float);

  float* d_output{nullptr};
  hipMalloc(&d_output, out_tensor_bytes);
  hipMemset(d_output, 0, out_tensor_bytes);

  int kernel_bytes = KernelDims.n * KernelDims.h * KernelDims.w  * KernelDims.c * sizeof(float);

  float* d_kernel{nullptr};
  hipMalloc(&d_kernel, kernel_bytes);
 

  timer.Start();

  hipMemcpy(d_input, h_input , in_tensor_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_kernel, h_kernels, kernel_bytes, hipMemcpyHostToDevice);
  timer.Stop(); 

  //printf("hipMemcpyHostToDevice:\t %f msecs.\n", timer.Elapsed());

  // The Convolution
  const float alpha = 1, beta = 0;

  timer.Start();
  checkCUDNN(hipdnnConvolutionForward(cudnn, &alpha, input_descriptor, d_input, kernel_descriptor, d_kernel, convolution_descriptor,
                                   convolution_algorithm, d_workspace, workspace_bytes, &beta, output_descriptor, d_output));
  timer.Stop(); 

  //printf("cuDNN Convolution:\t %f msecs.\n", timer.Elapsed());
 
  timer.Start();
  hipMemcpy(h_output, d_output, out_tensor_bytes, hipMemcpyDeviceToHost);
  timer.Stop();  

  //printf("hipMemcpyDeviceToHost:\t %f msecs.\n", timer.Elapsed());


  //delete[] h_output;
  hipFree(d_kernel);
  hipFree(d_input);
  hipFree(d_output);
  hipFree(d_workspace);

  hipdnnDestroyTensorDescriptor(input_descriptor);
  hipdnnDestroyTensorDescriptor(output_descriptor);
  hipdnnDestroyFilterDescriptor(kernel_descriptor);
  hipdnnDestroyConvolutionDescriptor(convolution_descriptor);

  hipdnnDestroy(cudnn); 
}


